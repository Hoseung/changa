#include "hip/hip_runtime.h"
#ifdef _WIN32
#define NOMINMAX
#endif

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>
// #include <cutil.h>

#include "hip/hip_runtime.h"
#include "CudaFunctions.h"

#define BLOCK_SIZE 64
#define PART_CACHE_SIZE 3

extern workRequestQueue *wrQueue;
extern void **devBuffers;
extern hipStream_t kernel_stream;

//__constant__ constantData[88];

void DataManagerTransfer(CudaMultipoleMoments *moments, int nMoments, CompactPartData *compactParts, int nCompactParts) {

	//int numBlocks = (int) ceilf((float)n/BLOCK_SIZE);

	//workRequest *transferKernel = (workRequest*) malloc(sizeof(workRequest));
	workRequest transferKernel;
	dataInfo *momentBuffer, *partCoreBuffer;

        // XXX - number of blocks is a dummy value since this
        // operation will not invoke a kernel
	transferKernel.dimGrid = dim3(1);
	transferKernel.dimBlock = dim3(BLOCK_SIZE);
	transferKernel.smemSize = 0;

	transferKernel.nBuffers = DM_TRANSFER_NBUFFERS;

	/* schedule two buffers for transfer to the GPU */
	transferKernel.bufferInfo = (dataInfo *) malloc(transferKernel.nBuffers * sizeof(dataInfo));

	momentBuffer = &(transferKernel.bufferInfo[POST_PREFETCH_MOMENTS]);
	momentBuffer->bufferID = POST_PREFETCH_MOMENTS;
	momentBuffer->transferToDevice = YES;
	momentBuffer->transferFromDevice = NO;
	momentBuffer->freeBuffer = NO;
	momentBuffer->hostBuffer = moments;
	momentBuffer->size = (nMoments) * sizeof(CudaMultipoleMoments);

	partCoreBuffer = &(transferKernel.bufferInfo[POST_PREFETCH_PARTICLE_CORES]);
	partCoreBuffer->bufferID = POST_PREFETCH_PARTICLE_CORES;
	partCoreBuffer->transferToDevice = YES;
	partCoreBuffer->transferFromDevice = NO;
	partCoreBuffer->freeBuffer = NO;
	partCoreBuffer->hostBuffer = compactParts;
	partCoreBuffer->size = (nCompactParts)*sizeof(CompactPartData);

	transferKernel.callbackFn = 0;
	transferKernel.id = DM_TRANSFER;
	enqueue(wrQueue, &transferKernel);

}

void TreePieceCellListDataTransferLocal(CudaRequest *data){
	int numBlocks = data->numBucketsPlusOne-1;

	workRequest gravityKernel;
	//dataInfo *buffer, *partCoreBuffer;

	gravityKernel.dimGrid = dim3(numBlocks);
	gravityKernel.dimBlock = dim3(BLOCK_SIZE);
	gravityKernel.smemSize = 0;

	gravityKernel.nBuffers = TP_GRAVITY_LOCAL_NBUFFERS;

	/* schedule buffers for transfer to the GPU */
	gravityKernel.bufferInfo = (dataInfo *) malloc(gravityKernel.nBuffers * sizeof(dataInfo));

	TreePieceCellListDataTransferBasic(data, &gravityKernel);

	gravityKernel.callbackFn = 0;
	gravityKernel.id = TP_GRAVITY_LOCAL;
	enqueue(wrQueue, &gravityKernel);
}

void TreePieceCellListDataTransferRemote(CudaRequest *data){
	int numBlocks = data->numBucketsPlusOne-1;

	workRequest gravityKernel;
	//dataInfo *buffer, *partCoreBuffer;

	gravityKernel.dimGrid = dim3(numBlocks);
	gravityKernel.dimBlock = dim3(BLOCK_SIZE);
	gravityKernel.smemSize = 0;

	gravityKernel.nBuffers = TP_NODE_GRAVITY_REMOTE_NBUFFERS;

	/* schedule buffers for transfer to the GPU */
	gravityKernel.bufferInfo = (dataInfo *) malloc(gravityKernel.nBuffers * sizeof(dataInfo));

	TreePieceCellListDataTransferBasic(data, &gravityKernel);

	gravityKernel.callbackFn = 0;
	gravityKernel.id = TP_GRAVITY_REMOTE;
	enqueue(wrQueue, &gravityKernel);
}

void TreePieceCellListDataTransferRemoteResume(CudaRequest *data, CudaMultipoleMoments *missedMoments, int numMissedMoments){
	int numBlocks = data->numBucketsPlusOne-1;

	//workRequest *gravityKernel = (workRequest*) malloc(sizeof(workRequest));
	workRequest gravityKernel;
	dataInfo *buffer;

	gravityKernel.dimGrid = dim3(numBlocks);
	gravityKernel.dimBlock = dim3(BLOCK_SIZE);
	gravityKernel.smemSize = 0;

	gravityKernel.nBuffers = TP_NODE_GRAVITY_REMOTE_RESUME_NBUFFERS;

	/* schedule buffers for transfer to the GPU */
	gravityKernel.bufferInfo = (dataInfo *) malloc(gravityKernel.nBuffers * sizeof(dataInfo));

	TreePieceCellListDataTransferBasic(data, &gravityKernel);

	buffer = &(gravityKernel.bufferInfo[MISSED_MOMENTS]);
	buffer->bufferID = MISSED_MOMENTS;
	buffer->transferToDevice = YES;
	buffer->transferFromDevice = NO;
	buffer->freeBuffer = YES;
	buffer->hostBuffer = missedMoments;
	buffer->size = (numMissedMoments) * sizeof(int);

	gravityKernel.callbackFn = 0;
	gravityKernel.id = TP_GRAVITY_REMOTE_RESUME;
	enqueue(wrQueue, &gravityKernel);
}

void TreePieceCellListDataTransferBasic(CudaRequest *data, workRequest *gravityKernel){
	dataInfo *buffer;
	int numBucketsPlusOne = data->numBucketsPlusOne;
        int numBuckets = numBucketsPlusOne-1;
        int size;

	buffer = &(gravityKernel->bufferInfo[ILCELL]);
	buffer->bufferID = ILCELL;
	buffer->transferToDevice = YES;
	buffer->transferFromDevice = NO;
	buffer->freeBuffer = YES;
        size = (data->numInteractions) * sizeof(ILCell);
	buffer->size = size;
        buffer->hostBuffer = malloc(size);
        memcpy(buffer->hostBuffer, data->list, size);

	buffer = &(gravityKernel->bufferInfo[NODE_BUCKET_MARKERS]);
	buffer->bufferID = NODE_BUCKET_MARKERS;
	buffer->transferToDevice = YES;
	buffer->transferFromDevice = NO;
	buffer->freeBuffer = YES;
        size = (numBucketsPlusOne) * sizeof(int);
        buffer->size = size;
        buffer->hostBuffer = malloc(size);
        memcpy(buffer->hostBuffer, data->bucketMarkers, size);

	buffer = &(gravityKernel->bufferInfo[NODE_BUCKET_START_MARKERS]);
	buffer->bufferID = NODE_BUCKET_START_MARKERS;
	buffer->transferToDevice = YES;
	buffer->transferFromDevice = NO;
	buffer->freeBuffer = YES;
	size = (numBuckets) * sizeof(int);
        buffer->size = size;
        buffer->hostBuffer = malloc(size);
        memcpy(buffer->hostBuffer, data->bucketStarts, size);

	buffer = &(gravityKernel->bufferInfo[NODE_BUCKET_SIZES]);
	buffer->bufferID = NODE_BUCKET_SIZES;
	buffer->transferToDevice = YES;
	buffer->transferFromDevice = NO;
	buffer->freeBuffer = YES;
        buffer->size = size;
        buffer->hostBuffer = malloc(size);
        memcpy(buffer->hostBuffer, data->bucketSizes, size);

        // copy affectedBuckets array
        int *save = data->affectedBuckets;
        data->affectedBuckets = (int *) malloc(sizeof(int)*numBuckets);
        memcpy(data->affectedBuckets, save, sizeof(int)*numBuckets);
}

void TreePiecePartListDataTransferLocal(CudaRequest *data){
	int numBlocks = data->numBucketsPlusOne-1;

	workRequest gravityKernel;
	//dataInfo *buffer, *partCoreBuffer;

	gravityKernel.dimGrid = dim3(numBlocks);
	gravityKernel.dimBlock = dim3(BLOCK_SIZE);
	gravityKernel.smemSize = 0;

	gravityKernel.nBuffers = TP_GRAVITY_LOCAL_NBUFFERS;

	/* schedule buffers for transfer to the GPU */
	gravityKernel.bufferInfo = (dataInfo *) malloc(gravityKernel.nBuffers * sizeof(dataInfo));

	TreePiecePartListDataTransferBasic(data, &gravityKernel);

	gravityKernel.callbackFn = 0;
	gravityKernel.id = TP_PART_GRAVITY_LOCAL;
	enqueue(wrQueue, &gravityKernel);
}

void TreePiecePartListDataTransferRemote(CudaRequest *data){
//void TreePiecePartListDataTransferRemote(PartListData *data, CompactPartData *missedParts, int numMissedParts){
	int numBlocks = data->numBucketsPlusOne-1;

	//workRequest *gravityKernel = (workRequest*) malloc(sizeof(workRequest));
	workRequest gravityKernel;
	//dataInfo *buffer;

	gravityKernel.dimGrid = dim3(numBlocks);
	gravityKernel.dimBlock = dim3(BLOCK_SIZE);
	gravityKernel.smemSize = 0;

	gravityKernel.nBuffers = TP_PART_GRAVITY_REMOTE_NBUFFERS;

	/* schedule buffers for transfer to the GPU */
	gravityKernel.bufferInfo = (dataInfo *) malloc(gravityKernel.nBuffers * sizeof(dataInfo));

	TreePiecePartListDataTransferBasic(data, &gravityKernel);

/*
	buffer = &(gravityKernel.bufferInfo[MISSED_PARTS]);
	buffer->bufferID = MISSED_PARTS;
	buffer->transferToDevice = YES;
	buffer->transferFromDevice = NO;
	buffer->freeBuffer = YES;
	buffer->hostBuffer = missedParts;
	buffer->size = (numMissedParts) * sizeof(int);
*/

	gravityKernel.callbackFn = 0;
	gravityKernel.id = TP_PART_GRAVITY_REMOTE;
	enqueue(wrQueue, &gravityKernel);
}

void TreePiecePartListDataTransferRemoteResume(CudaRequest *data, CompactPartData *missedParts, int numMissedParts){
	int numBlocks = data->numBucketsPlusOne-1;

	//workRequest *gravityKernel = (workRequest*) malloc(sizeof(workRequest));
	workRequest gravityKernel;
	dataInfo *buffer;

	gravityKernel.dimGrid = dim3(numBlocks);
	gravityKernel.dimBlock = dim3(BLOCK_SIZE);
	gravityKernel.smemSize = 0;

	gravityKernel.nBuffers = TP_PART_GRAVITY_REMOTE_RESUME_NBUFFERS;

	/* schedule buffers for transfer to the GPU */
	gravityKernel.bufferInfo = (dataInfo *) malloc(gravityKernel.nBuffers * sizeof(dataInfo));

	TreePiecePartListDataTransferBasic(data, &gravityKernel);

	buffer = &(gravityKernel.bufferInfo[MISSED_PARTS]);
	buffer->bufferID = MISSED_PARTS;
	buffer->transferToDevice = YES;
	buffer->transferFromDevice = NO;
	buffer->freeBuffer = YES;
	buffer->hostBuffer = missedParts;
	buffer->size = (numMissedParts) * sizeof(int);

	gravityKernel.callbackFn = 0;
	gravityKernel.id = TP_PART_GRAVITY_REMOTE_RESUME;
	enqueue(wrQueue, &gravityKernel);
}

void TreePiecePartListDataTransferBasic(CudaRequest *data, workRequest *gravityKernel){
	dataInfo *buffer;

	int numInteractions = data->numInteractions;
	int numBucketsPlusOne = data->numBucketsPlusOne;
        int numBuckets = numBucketsPlusOne-1;
        int size;

	buffer = &(gravityKernel->bufferInfo[ILPART]);
	buffer->bufferID = ILPART;
	buffer->transferToDevice = YES;
	buffer->transferFromDevice = NO;
	buffer->freeBuffer = YES;
	size = (numInteractions) * sizeof(ILPart);
        buffer->size = size;
        buffer->hostBuffer = malloc(size);
        memcpy(buffer->hostBuffer, data->list, size);


	buffer = &(gravityKernel->bufferInfo[PART_BUCKET_MARKERS]);
	buffer->bufferID = PART_BUCKET_MARKERS;
	buffer->transferToDevice = YES;
	buffer->transferFromDevice = NO;
	buffer->freeBuffer = YES;
	size = (numBucketsPlusOne) * sizeof(int);
        buffer->size = size;
        buffer->hostBuffer = malloc(size);
        memcpy(buffer->hostBuffer, data->bucketMarkers, size);

	buffer = &(gravityKernel->bufferInfo[PART_BUCKET_START_MARKERS]);
	buffer->bufferID = PART_BUCKET_START_MARKERS;
	buffer->transferToDevice = YES;
	buffer->transferFromDevice = NO;
	buffer->freeBuffer = YES;
	buffer->hostBuffer = data->bucketStarts;
	size = (numBuckets) * sizeof(int);
        buffer->size = size;
        buffer->hostBuffer = malloc(size);
        memcpy(buffer->hostBuffer, data->bucketStarts, size);

	buffer = &(gravityKernel->bufferInfo[PART_BUCKET_SIZES]);
	buffer->bufferID = PART_BUCKET_SIZES;
	buffer->transferToDevice = YES;
	buffer->transferFromDevice = NO;
	buffer->freeBuffer = YES;
        buffer->size = size;
        buffer->hostBuffer = malloc(size);
        memcpy(buffer->hostBuffer, data->bucketSizes, size);

        // copy affectedBuckets array
        int *save = data->affectedBuckets;
        data->affectedBuckets = (int *)malloc(sizeof(int)*numBuckets);
        memcpy(data->affectedBuckets, save, sizeof(int)*numBuckets);
}

/*
__global__ void EwaldTopKernel(GravityParticleData *particleTable) {
*/

/* kernels:

TP_GRAVITY_LOCAL,
TP_GRAVITY_REMOTE,
TP_GRAVITY_REMOTE_RESUME,
TP_PART_GRAVITY_LOCAL,
TP_PART_GRAVITY_REMOTE,
TP_PART_GRAVITY_REMOTE_RESUME

 */

extern void DeleteHostMoments(CudaMultipoleMoments *array);
extern void DeleteHostParticles(CompactPartData *array);
static boolean dmTransferDone = NO;

void FreeDataManagerMemory(){
  workRequest gravityKernel;
  dataInfo *buffer;

  gravityKernel.dimGrid = dim3(1);
  gravityKernel.dimBlock = dim3(BLOCK_SIZE);
  gravityKernel.smemSize = 0;

  gravityKernel.nBuffers = DM_TRANSFER_NBUFFERS ;

  /* schedule buffers for transfer to the GPU */
  gravityKernel.bufferInfo = (dataInfo *) malloc(DM_TRANSFER_NBUFFERS * sizeof(dataInfo));

  buffer = &(gravityKernel.bufferInfo[POST_PREFETCH_MOMENTS]);
  buffer->bufferID = POST_PREFETCH_MOMENTS;
  buffer->transferToDevice = NO ;
  buffer->transferFromDevice = NO;
  buffer->freeBuffer = YES;
  buffer->hostBuffer = 0;
  // FIXME - need actual size?
  buffer->size = 0 

  buffer = &(gravityKernel.bufferInfo[POST_PREFETCH_PARTICLE_CORES]);
  buffer->bufferID = POST_PREFETCH_PARTICLE_CORES;
  buffer->transferToDevice = NO ;
  buffer->transferFromDevice = NO;
  buffer->freeBuffer = YES;
  buffer->hostBuffer = 0;
  // FIXME - need actual size?
  buffer->size = 0 

  gravityKernel.callbackFn = 0;
  gravityKernel.id = DM_TRANSFER;
  enqueue(wrQueue, &gravityKernel);
}

// kernel selector function
void kernelSelect(workRequest *wr) {

  switch (wr->id) {
  case DM_TRANSFER:
	  dmTransferDone = YES;
	  DeleteHostMoments((CudaMultipoleMoments *)wr->bufferInfo[POST_PREFETCH_MOMENTS].hostBuffer);
	  DeleteHostParticles((CompactPartData *)wr->bufferInfo[POST_PREFETCH_PARTICLE_CORES].hostBuffer);
	  break;

  case TP_GRAVITY_LOCAL:
	  // FIXME - fix arguments
	  if(dmTransferDone){
		  //GravityKernel<<<wr->dimGrid, wr->dimBlock, wr->smemSize, kernel_stream>>>
		  //((GravityParticleData *)devBuffers[wr->bufferInfo[PARTICLE_TABLE].bufferID],
			//(EwtData *)devBuffers[wr->bufferInfo[EWALD_TABLE].bufferID]);

                  // delete arrays allocated earlier
                  free((ILCell *)wr->bufferInfo[ILCELL].hostBuffer);
                  free((int *)wr->bufferInfo[NODE_BUCKET_MARKERS].hostBuffer);
                  free((int *)wr->bufferInfo[NODE_BUCKET_START_MARKERS].hostBuffer);
                  free((int *)wr->bufferInfo[NODE_BUCKET_SIZES].hostBuffer);
	  }
	  else{
		  // fix buffer transfer flags and re-enqueue
		  NoTransferEnqueueNodeBasic(wr);
	  }
    break;

  case TP_PART_GRAVITY_LOCAL:
  	  // FIXME - fix arguments
  	  if(dmTransferDone){
  		  //GravityKernel<<<wr->dimGrid, wr->dimBlock, wr->smemSize, kernel_stream>>>
  		  //((GravityParticleData *)devBuffers[wr->bufferInfo[PARTICLE_TABLE].bufferID],
  			//(EwtData *)devBuffers[wr->bufferInfo[EWALD_TABLE].bufferID]);
                  // delete arrays allocated earlier
                  free((ILPart *)wr->bufferInfo[ILPART].hostBuffer);
                  free((int *)wr->bufferInfo[PART_BUCKET_MARKERS].hostBuffer);
                  free((int *)wr->bufferInfo[PART_BUCKET_START_MARKERS].hostBuffer);
                  free((int *)wr->bufferInfo[PART_BUCKET_SIZES].hostBuffer);
  	  }
  	  else{
  		  // fix buffer transfer flags and re-enqueue
  		  NoTransferEnqueuePartBasic(wr);
  	  }
      break;

  case TP_GRAVITY_REMOTE:
	  if(dmTransferDone){
		  // invoke kernel
	  }
	  else{
		  // fix buffer transfer flags and re-enqueue
		  NoTransferEnqueueNodeBasic(wr);
	  }
	  break;

  case TP_PART_GRAVITY_REMOTE:
	  if(dmTransferDone){
		  // invoke kernel
	  }
	  else{
		  // fix buffer transfer flags and re-enqueue
		  NoTransferEnqueuePartBasic(wr);

	  }
	  break;

	  // FIXME - remote resume for parts and nodes
	  // after that, set callbacks
	  // finally, do book-keeping, remaining mindful of the lastBucketComplete flag in the state

  default:
    printf("error: id %d not valid\n", wr->id);
    break;
  }
}

void NoTransferEnqueueNodeBasic(workRequest *gravityKernel){
	dataInfo *buffer;

	// all buffer transfer parameters set to NO because there already exists a workRequest that will do all
	// of this
	workRequest *newGravityKernel = (workRequest*) malloc(sizeof(workRequest));
	*newGravityKernel = *gravityKernel;

	buffer = &(newGravityKernel->bufferInfo[ILCELL]);
	buffer->transferToDevice = NO;
	buffer->transferFromDevice = NO;
	buffer->freeBuffer = NO;

	buffer = &(newGravityKernel->bufferInfo[NODE_BUCKET_MARKERS]);
	buffer->transferToDevice = NO;
	buffer->transferFromDevice = NO;
	buffer->freeBuffer = NO;

	buffer = &(newGravityKernel->bufferInfo[NODE_BUCKET_START_MARKERS]);
	buffer->transferToDevice = NO;
	buffer->transferFromDevice = NO;
	buffer->freeBuffer = NO;

	buffer = &(newGravityKernel->bufferInfo[NODE_BUCKET_SIZES]);
	buffer->transferToDevice = NO;
	buffer->transferFromDevice = NO;
	buffer->freeBuffer = NO;

	enqueue(wrQueue, newGravityKernel);
}

void NoTransferEnqueuePartBasic(workRequest *gravityKernel){
	dataInfo *buffer;

	// all buffer transfer parameters set to NO because there already exists a workRequest that will do all
	// of this
	workRequest *newGravityKernel = (workRequest*) malloc(sizeof(workRequest));
	*newGravityKernel = *gravityKernel;

	buffer = &(newGravityKernel->bufferInfo[ILPART]);
	buffer->transferToDevice = NO;
	buffer->transferFromDevice = NO;
	buffer->freeBuffer = NO;

	buffer = &(newGravityKernel->bufferInfo[PART_BUCKET_MARKERS]);
	buffer->transferToDevice = NO;
	buffer->transferFromDevice = NO;
	buffer->freeBuffer = NO;

	buffer = &(newGravityKernel->bufferInfo[PART_BUCKET_START_MARKERS]);
	buffer->transferToDevice = NO;
	buffer->transferFromDevice = NO;
	buffer->freeBuffer = NO;

	buffer = &(newGravityKernel->bufferInfo[PART_BUCKET_SIZES]);
	buffer->transferToDevice = NO;
	buffer->transferFromDevice = NO;
	buffer->freeBuffer = NO;

	enqueue(wrQueue, gravityKernel);
}

/*
 * Kernels
 */

#define GROUP(t)  ((t)/MAX_THREADS_PER_GROUP)
#define GROUP_INDEX(t) ((t)%MAX_THREADS_PER_GROUP)

__global__ void nodeGravityComputation(
		CompactPartData *particleCores,
		VariablePartData *particleVars,
		CudaMultipoleMoments *moments,
		ILCell *ils,
		int numInteractions,
		int *ilmarks,
		int *bucketStarts,
		int *bucketSizes,
		int numBucketsPlusOne, cudatype fperiod){

  // each thread has its own storage for these
  __shared__ CudaVector3D acc[THREADS_PER_BLOCK];
  __shared__ cudatype pot[THREADS_PER_BLOCK];
  __shared__ CudaMultipoleMoments m[THREADS_PER_BLOCK];

  // to store a few particles in shared memory 
  __shared__ CompactPartData cached_particle_cores[PART_CACHE_SIZE];
  // in case PART_CACHE_SIZE < bucketSize, need this extra
  __shared__ CompactPartData shared_particle_core;


  // each block is given a bucket to compute
  // each thread in the block computes an interaction of a particle with a node
  // threads must iterate through the interaction lists and sync.
  // then, block leader (first rank in each block) reduces the forces and commits 
  // values to global memory.
  int bucket = blockIdx.x;
  int start = ilmarks[bucket];
  int end = ilmarks[bucket+1];
  int bucketSize = bucketSizes[bucket];
  int bucketStart = bucketStarts[bucket];
  int thread = threadIdx.x;

  // length of cell interaction list for this bucket
  int llen = end-start;

  CudaVector3D r;
  cudatype rsq;
  cudatype twoh, a, b, c, d;

#ifdef __DEVICE_EMULATION__
  if(blockIdx.x == 0){
    //printf("t: %d, blen: %d, llen: %d\n", threadIdx.x, blen, llen);
    //printf("group: %d, particle: %d, ngroups: %d, groupSize: %d\n", group, particle, ngroups, groupSize);
  }
#endif

    // get some threads to load particles into part_cache
  if(thread < PART_CACHE_SIZE){
    cached_particle_cores[thread] = partCores[bucketStart+thread];
  }
  __syncthreads();


  for(int node = thread; node < llen; node += THREADS_PER_BLOCK){
#ifdef __DEVICE_EMULATION__
    if(blockIdx.x == 0){
      //printf("t: %d, particle: %d, node: %d\n", threadIdx.x, particle, node);
      //printf("shared_moments[%d] = moments[%d]\n", threadIdx.x, ils[ilmarks[bucket]+node].index);
    }
#endif
    // FIXME - ought to keep ilmarks[blockIdx.x] in a register
    // all threads in a group access different mem locations
    // however, threads accessing the same node in different
    // groups access the same memory location
    // therefore, there ought to be 8 accesses for every 4 groups
    // FIXME - can this be reduced somehow?

    m[thread] = moments[ils[ilmarks[bucket]+node].index];
    int offsetID = ils[ilmarks[bucket]+node].offsetID;

    __syncthreads();

    for(int particle = 0; particle < bucketSize; particle++){
      
      acc[thread].x = 0;
      acc[thread].y = 0;
      acc[thread].z = 0;
      pot[thread] = 0;
      
      // true or false for all threads in block, so no divergence
      if(particle < PART_CACHE_SIZE){
        // use cached_particle_cores
          
        r.x = cached_particle_cores[particle].position.x -
                        ((((offsetID >> 22) & 0x7)-3)*fperiod + m[thread].cm.x);
        r.y = cached_particle_cores[particle].position.y -
                        ((((offsetID >> 25) & 0x7)-3)*fperiod + m[thread].cm.y);
        r.z = cached_particle_cores[particle].position.z -
                        ((((offsetID >> 28) & 0x7)-3)*fperiod + m[thread].cm.z);

        rsq = r.x*r.x + r.y*r.y + r.z*r.z;        
        twoh = m[thread].soft + cached_particle_cores[particle].soft;
        if(rsq != 0){
          cudatype dir = 1.0/sqrt(rsq);
          // SPLINEQ(dir, rsq, twoh, a, b, c, d);
          // expansion of function below:
          cudatype u,dih;
          if (rsq < twoh*twoh) {
            dih = 2.0/twoh;
            u = dih/dir;
            if (u < 1.0) {
              a = dih*(7.0/5.0 - 2.0/3.0*u*u + 3.0/10.0*u*u*u*u
            		 - 1.0/10.0*u*u*u*u*u);
              b = dih*dih*dih*(4.0/3.0 - 6.0/5.0*u*u + 1.0/2.0*u*u*u);
              c = dih*dih*dih*dih*dih*(12.0/5.0 - 3.0/2.0*u);
              d = 3.0/2.0*dih*dih*dih*dih*dih*dih*dir;
            }
            else {
              a = -1.0/15.0*dir + dih*(8.0/5.0 - 4.0/3.0*u*u + u*u*u
                          - 3.0/10.0*u*u*u*u + 1.0/30.0*u*u*u*u*u);
              b = -1.0/15.0*dir*dir*dir + dih*dih*dih*(8.0/3.0 - 3.0*u
                          + 6.0/5.0*u*u - 1.0/6.0*u*u*u);
              c = -1.0/5.0*dir*dir*dir*dir*dir + 3.0*dih*dih*dih*dih*dir
            	+ dih*dih*dih*dih*dih*(-12.0/5.0 + 1.0/2.0*u);
              d = -dir*dir*dir*dir*dir*dir*dir
            	+ 3.0*dih*dih*dih*dih*dir*dir*dir
            	- 1.0/2.0*dih*dih*dih*dih*dih*dih*dir;
            }
          }
          else {
            a = dir;
            b = a*a*a;
            c = 3.0*b*a*a;
            d = 5.0*c*a*a;
          }
     
          cudatype qirx = m[thread].xx*r.x + m[thread].xy*r.y + m[thread].xz*r.z;
          cudatype qiry = m[thread].xy*r.x + m[thread].yy*r.y + m[thread].yz*r.z;
          cudatype qirz = m[thread].xz*r.x + m[thread].yz*r.y + m[thread].zz*r.z;
          cudatype qir = 0.5*(qirx*r.x + qiry*r.y + qirz*r.z);
          cudatype tr = 0.5*(m[thread].xx + m[thread].yy + m[thread].zz);
          cudatype qir3 = b*m[thread].totalMass + d*qir - c*tr;

          pot[thread] -= m[thread].totalMass * a + c*qir - b*tr;

          acc[thread].x -= qir3*r.x - c*qirx;
          acc[thread].y -= qir3*r.y - c*qiry;
          acc[thread].z -= qir3*r.z - c*qirz;
        }
      }
      else{
        if(thread == 0){
          // load shared_particle_core and use
          shared_particle_core = particleCores[bucketStart+particle];
          // FIXME - where is the initial particle core load in 
          // the previous case (particle < PART_CACHE_SIZE) ?
        }
        __syncthreads();
          
        r.x = shared_particle_core.position.x -
                        ((((offsetID >> 22) & 0x7)-3)*fperiod + m[thread].cm.x);
        r.y = shared_particle_core.position.y -
                        ((((offsetID >> 25) & 0x7)-3)*fperiod + m[thread].cm.y);
        r.z = shared_particle_core.position.z -
                        ((((offsetID >> 28) & 0x7)-3)*fperiod + m[thread].cm.z);

        rsq = r.x*r.x + r.y*r.y + r.z*r.z;        
        twoh = m[thread].soft + shared_particle_core.soft;
        if(rsq != 0){
          cudatype dir = 1.0/sqrt(rsq);
          // SPLINEQ(dir, rsq, twoh, a, b, c, d);
          // expansion of function below:
          cudatype u,dih;
          if (rsq < twoh*twoh) {
            dih = 2.0/twoh;
            u = dih/dir;
            if (u < 1.0) {
              a = dih*(7.0/5.0 - 2.0/3.0*u*u + 3.0/10.0*u*u*u*u
            		 - 1.0/10.0*u*u*u*u*u);
              b = dih*dih*dih*(4.0/3.0 - 6.0/5.0*u*u + 1.0/2.0*u*u*u);
              c = dih*dih*dih*dih*dih*(12.0/5.0 - 3.0/2.0*u);
              d = 3.0/2.0*dih*dih*dih*dih*dih*dih*dir;
            }
            else {
              a = -1.0/15.0*dir + dih*(8.0/5.0 - 4.0/3.0*u*u + u*u*u
                          - 3.0/10.0*u*u*u*u + 1.0/30.0*u*u*u*u*u);
              b = -1.0/15.0*dir*dir*dir + dih*dih*dih*(8.0/3.0 - 3.0*u
                          + 6.0/5.0*u*u - 1.0/6.0*u*u*u);
              c = -1.0/5.0*dir*dir*dir*dir*dir + 3.0*dih*dih*dih*dih*dir
            	+ dih*dih*dih*dih*dih*(-12.0/5.0 + 1.0/2.0*u);
              d = -dir*dir*dir*dir*dir*dir*dir
            	+ 3.0*dih*dih*dih*dih*dir*dir*dir
            	- 1.0/2.0*dih*dih*dih*dih*dih*dih*dir;
            }
          }
          else {
            a = dir;
            b = a*a*a;
            c = 3.0*b*a*a;
            d = 5.0*c*a*a;
          }
     
          cudatype qirx = m[thread].xx*r.x + m[thread].xy*r.y + m[thread].xz*r.z;
          cudatype qiry = m[thread].xy*r.x + m[thread].yy*r.y + m[thread].yz*r.z;
          cudatype qirz = m[thread].xz*r.x + m[thread].yz*r.y + m[thread].zz*r.z;
          cudatype qir = 0.5*(qirx*r.x + qiry*r.y + qirz*r.z);
          cudatype tr = 0.5*(m[thread].xx + m[thread].yy + m[thread].zz);
          cudatype qir3 = b*m[thread].totalMass + d*qir - c*tr;

          pot[thread] -= m[thread].totalMass * a + c*qir - b*tr;

          acc[thread].x -= qir3*r.x - c*qirx;
          acc[thread].y -= qir3*r.y - c*qiry;
          acc[thread].z -= qir3*r.z - c*qirz;
        }// end if rsq != 0
      }// end else (particle >= PART_CACHE_SIZE)
      
      __syncthreads();
      // all threads have computed their portion of the forces
      // time to add forces up
      
      cudatype sum = 0.0;
      if(thread == 0){
        for(int i = 0; i < THREADS_PER_BLOCK; i++){
          sum += acc[i].x;
        }
        particleVars[bucketStart+particle].a.x += sum;
        
        sum = 0;
        for(int i = 0; i < THREADS_PER_BLOCK; i++){
          sum += acc[i].y;
        }
        particleVars[bucketStart+particle].a.y += sum;

        sum = 0;
        for(int i = 0; i < THREADS_PER_BLOCK; i++){
          sum += acc[i].z;
        }
        particleVars[bucketStart+particle].a.z += sum;
        
        sum = 0;
        for(int i = 0; i < THREADS_PER_BLOCK; i++){
          sum += pot[i];
        }
        particleVars[bucketStart+particle].potential += sum;
      }
    }// end for each particle
  }// end for each thread (node)
}

__global__ void particleGravityComputation(
                                   CompactPartData *particleCores,
                                   VariablePartData *particleVars,
                                   ILPart *ils,
		                   int numInteractions,
                                   int *ilmarks,
		                   int *bucketStarts,
		                   int *bucketSizes,
		                   int numBucketsPlusOne, cudatype fperiod){

  // each thread has its own storage for these
  __shared__ CudaVector3D acc[THREADS_PER_BLOCK];
  __shared__ cudatype pot[THREADS_PER_BLOCK];
  __shared__ CompactPartData source_cores[THREADS_PER_BLOCK];

  // to store a few particles in shared memory 
  __shared__ CompactPartData cached_target_cores[PART_CACHE_SIZE];
  // in case PART_CACHE_SIZE < bucketSize, need this extra
  __shared__ CompactPartData shared_target_core;


  // each block is given a bucket to compute
  // each thread in the block computes an interaction of a particle with a node
  // threads must iterate through the interaction lists and sync.
  // then, block leader (first rank in each block) reduces the forces and commits 
  // values to global memory.
  int bucket = blockIdx.x;
  int start = ilmarks[bucket];
  int end = ilmarks[bucket+1];
  int bucketSize = bucketSizes[bucket];
  int bucketStart = bucketStarts[bucket];
  int thread = threadIdx.x;

  // length of cell interaction list for this bucket
  int llen = end-start;

  CudaVector3D r;
  cudatype rsq;
  cudatype twoh, a, b;

#ifdef __DEVICE_EMULATION__
  if(blockIdx.x == 0){
    //printf("t: %d, blen: %d, llen: %d\n", threadIdx.x, blen, llen);
    //printf("group: %d, particle: %d, ngroups: %d, groupSize: %d\n", group, particle, ngroups, groupSize);
  }
#endif

  // get some threads to load particles into part_cache
  if(thread < PART_CACHE_SIZE){
    cached_particle_cores[thread] = partCores[bucketStart+thread];
  }
  __syncthreads();


  for(int source = thread; source < llen; source += THREADS_PER_BLOCK){
#ifdef __DEVICE_EMULATION__
    if(blockIdx.x == 0){
      //printf("t: %d, particle: %d, node: %d\n", threadIdx.x, particle, node);
      //printf("shared_moments[%d] = moments[%d]\n", threadIdx.x, ils[ilmarks[bucket]+node].index);
    }
#endif

    source_cores[thread] = particleCores[ils[ilmarks[bucket]+source].index];
    int oid = ils[ilmarks[bucket]+source].off;

    for(int target = 0; target < bucketSize; target++){
      
      acc[thread].x = 0;
      acc[thread].y = 0;
      acc[thread].z = 0;
      pot[thread] = 0;
      
      // true or false for all threads in block, so no divergence
      if(target < PART_CACHE_SIZE){
        // use cached_particle_cores
        r.x = (((oid >> 22) & 0x7)-3)*fperiod +
                source_cores[thread].position.x -
                cached_target_cores[target].position.x;

        r.y = (((oid >> 25) & 0x7)-3)*fperiod +
                source_cores[thread].position.y -
                cached_target_cores[target].position.y;

        r.z = (((oid >> 28) & 0x7)-3)*fperiod +
                source_cores[thread].position.z -
                cached_target_cores[target].position.z;

        rsq = r.x*r.x + r.y*r.y + r.z*r.z;
        twoh = source_cores[thread].soft + cached_target_cores[target].soft;
        if(rsq != 0){
          //SPLINE(rsq, twoh, a, b);
          //SPLINE(r2, twoh, a, b);
          //expanded below:
	  cudatype r1, u,dih,dir;
	  r1 = sqrt(rsq);
	  if (r1 < (twoh)) {
		dih = 2.0/(twoh);
		u = r1*dih;
		if (u < 1.0) {
			a = dih*(7.0/5.0 - 2.0/3.0*u*u + 3.0/10.0*u*u*u*u
					 - 1.0/10.0*u*u*u*u*u);
			b = dih*dih*dih*(4.0/3.0 - 6.0/5.0*u*u
                                          + 1.0/2.0*u*u*u);
		}
		else {
			dir = 1.0/r1;
			a = -1.0/15.0*dir + dih*(8.0/5.0 - 4.0/3.0*u*u +
                            u*u*u - 3.0/10.0*u*u*u*u + 1.0/30.0*u*u*u*u*u);
			b = -1.0/15.0*dir*dir*dir +
                                dih*dih*dih*(8.0/3.0 - 3.0*u +
                                6.0/5.0*u*u - 1.0/6.0*u*u*u);
		}
	  }
	  else {
		a = 1.0/r1;
		b = a*a*a;
	  }
     
          pot[threadIdx.x] -= source_cores[thread].mass * a;

          acc[thread].x += r.x*b*source_cores[thread].mass;
          acc[thread].y += r.y*b*source_cores[thread].mass;
          acc[thread].z += r.z*b*source_cores[thread].mass;
        }
      }
      else{
        if(thread == 0){
          // load shared_particle_core and use
          shared_target_core = particleCores[bucketStart+target];
        }
        // use shared_target_core
        r.x = (((oid >> 22) & 0x7)-3)*fperiod +
                source_cores[thread].position.x -
                shared_target_core.position.x;

        r.y = (((oid >> 25) & 0x7)-3)*fperiod +
                source_cores[thread].position.y -
                shared_target_core.position.y;

        r.z = (((oid >> 28) & 0x7)-3)*fperiod +
                source_cores[thread].position.z -
                shared_target_core.position.z;

        rsq = r.x*r.x + r.y*r.y + r.z*r.z;
        twoh = source_cores[thread].soft + shared_target_core.soft;
        if(rsq != 0){
          //SPLINE(rsq, twoh, a, b);
          //SPLINE(r2, twoh, a, b);
          //expanded below:
	  cudatype r1, u,dih,dir;
	  r1 = sqrt(rsq);
	  if (r1 < (twoh)) {
		dih = 2.0/(twoh);
		u = r1*dih;
		if (u < 1.0) {
			a = dih*(7.0/5.0 - 2.0/3.0*u*u + 3.0/10.0*u*u*u*u
					 - 1.0/10.0*u*u*u*u*u);
			b = dih*dih*dih*(4.0/3.0 - 6.0/5.0*u*u
                                          + 1.0/2.0*u*u*u);
		}
		else {
			dir = 1.0/r1;
			a = -1.0/15.0*dir + dih*(8.0/5.0 - 4.0/3.0*u*u +
                            u*u*u - 3.0/10.0*u*u*u*u + 1.0/30.0*u*u*u*u*u);
			b = -1.0/15.0*dir*dir*dir +
                                dih*dih*dih*(8.0/3.0 - 3.0*u +
                                6.0/5.0*u*u - 1.0/6.0*u*u*u);
		}
	  }
	  else {
		a = 1.0/r1;
		b = a*a*a;
	  }
     
          pot[threadIdx.x] -= source_cores[thread].mass * a;

          acc[thread].x += r.x*b*source_cores[thread].mass;
          acc[thread].y += r.y*b*source_cores[thread].mass;
          acc[thread].z += r.z*b*source_cores[thread].mass;
        }
          
      }// end else (target >= PART_CACHE_SIZE)
      
      __syncthreads();
      // all threads have computed their portion of the forces
      // time to add forces up
      
      cudatype sum = 0.0;
      if(thread == 0){
        for(int i = 0; i < THREADS_PER_BLOCK; i++){
          sum += acc[i].x;
        }
        particleVars[bucketStart+target].a.x += sum;
        
        sum = 0;
        for(int i = 0; i < THREADS_PER_BLOCK; i++){
          sum += acc[i].y;
        }
        particleVars[bucketStart+target].a.y += sum;

        sum = 0;
        for(int i = 0; i < THREADS_PER_BLOCK; i++){
          sum += acc[i].z;
        }
        particleVars[bucketStart+target].a.z += sum;
        
        sum = 0;
        for(int i = 0; i < THREADS_PER_BLOCK; i++){
          sum += pot[i];
        }
        particleVars[bucketStart+target].potential += sum;
      }
    }// end for each target
  }// end for each thread (source)
}

